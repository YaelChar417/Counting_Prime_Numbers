/*============================================================================*/
/*======================== Autor: Yael Charles Marin =========================*/
/*======================= Fecha de entrega: 12/06/2025 =======================*/
/*=========================== Matricula: A01711111 ===========================*/
/*========================= Solucion: multihilo CUDA =========================*/
/*============== Para compilar: !nvcc -arch=sm_75 -o app cuda.cu =============*/
/*========================== Speed Up: 51,541.77778 ==========================*/
/*============================ Eficiencia: 314.59% ===========================*/
/*============================================================================*/
#include <iostream>
#include <math.h>
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

#define SIZE 5000000 // 5x10^6
#define REPS 10 // numero de repeticiones del programa
#define THREADS 512
#define BLOCKS min(32, ((SIZE / THREADS) + 1))

/*==============================================================================
*
* Función que recibe un número para determinar si es primo o no.
* Si el número es primo, devuelve true, si no, devuelve false.
* al ser __device__ puede ser llamada para ejecutarse en la GPU
*
* @param key -> int: número a verificar si es primo o no
* @return bool: true si el número es primo, false si no
*
==============================================================================*/
__device__ bool isPrime(int key){
    if(key < 2)
    {
        return false;
    }

    int limit = floor(sqrtf(key));

    for(int i = 2; i <= limit; i++)
    {
        if(key % i == 0)
        {
            return false;
        }
    }

    return true;
}

/*==============================================================================
*
* Función que distribuye conocer los números primos y sumarlos del array a todos
* los hilos, para después guardar el resultado parcial en una memoria compartida
* y esperar a que los hilos terminen su tarea para sumar todos los subtotales
* de cada bloque y almacenarlo en results
*
* @param *array -> int: array con los números a sumar
* @param *result -> long long: array donde se almacenan los resultados parciales
* @return void
*
==============================================================================*/
__global__ void sum(int *array, long long *results){
    __shared__ long long cache[THREADS];
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    long long acum = 0;

    while(index < SIZE)
    {
        if(isPrime(array[index]))
        {
            acum += array[index];
        }
        index += blockDim.x * gridDim.x;
    }

    cache[threadIdx.x] = acum;
    __syncthreads();

    int i = blockDim.x / 2;
    while (i > 0) 
    {
        if (threadIdx.x < i) 
        {
            cache[threadIdx.x] += cache[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (threadIdx.x == 0) 
    {
        results[blockIdx.x] = cache[0];
    }
}

/*==============================================================================
*
* Función que recibe un array y su tamaño para llenarlos de forma consecutiva
* desde 1 hasta el tamaño del array
*
* @param *array -> int: array con los números a sumar
* @param size -> int: tamaño del arreglo
* @return void 
* @complejidad_asintotica: O(n)
*
==============================================================================*/
void fillArray(int *array, int size){
    for(int i = 0; i < size; i++)
    {
        array[i] = i + 1;
    }
}

int main(){
    // Declaración de variables
    int *array;
    int *deviceArray;
    long long *deviceResults, result = 0;
    long long *results = new long long[BLOCKS];

    // Inicializar el array
    array = new int[SIZE];

    // variables usadas para medir el tiempo
    high_resolution_clock::time_point startTime, endTime;
    double time;

    // llenar el array del host desde 1 hasta size
    fillArray(array, SIZE);

    // reservar espacio de memoria en device para array y results
    hipMalloc((void**)&deviceArray, SIZE * sizeof(int));
    hipMalloc((void**)&deviceResults, BLOCKS * sizeof(long long));

    // Copiar del host a device el array con los numeros a sumar
    hipMemcpy(deviceArray, array, SIZE * sizeof(int), hipMemcpyHostToDevice);

    cout << "Comenzando...\n";
    time = 0.0;

    // ejecutar el programa 10 veces y comenzar a medir
    for(int j = 0; j < REPS; j++)
    {
        startTime = high_resolution_clock::now();

        // mandar a llamar la tarea global de sumar
        sum <<< BLOCKS, THREADS >>>(deviceArray, deviceResults);

        // Fin de la medición
        endTime = high_resolution_clock::now();
        time += duration<double, std::milli>(endTime - startTime).count();
    }

    // Pasar los resultados parciales del device al host
    hipMemcpy(results, deviceResults, BLOCKS * sizeof(long long), 
                hipMemcpyDeviceToHost);

    // sumar los resultados parciales para obtener el resultado final
    for (int i = 0; i < BLOCKS; i++) {
        result += results[i];
    }

    // imprimir el resultado y promedio del tiempo
    cout << "result = " << result << "\n";
    cout << "avg time = " << fixed << setprecision(3) 
         << (time / REPS) <<  " ms\n";

    // liberar memoria dinamica
    delete[] array;
    delete[] results;
    hipFree(deviceArray);
    hipFree(deviceResults);

    return 0;
}